/*	now let's drop some CUDA in there!
 *	instead of running all on the CPU, we will hand off the task of adding floats to the GPU
 *	for now, we'll stick with one thread
 *	compile with cuda compiler:
		nvcc cuda1.cu -o cuda1
*/
//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@//


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

int arraySize = 1<<20; // basically a million

// function to add them together
__global__																/*/*/	// this makes addArrays() accessible to the GPU
void addArrays (int arraySize, float *add1, float *add2, float *sum){			// addArrays() is now considered a kernel
	for (int i=0; i<arraySize; i++){
		sum[i] = add1[i] + add2[i];
	}
}

// all the action
int main(){

	// three arrays; we will add the first two to sum[]
	printf("initializing arrays\n");
	float *add1, *add2, *sum;											/*/*/	// CUDA allows us to set up a memory space
	hipMallocManaged( &add1, arraySize*sizeof(float) );				/*/*/	// accessible by the CPU and GPU alike
	hipMallocManaged( &add2, arraySize*sizeof(float) );				/*/*/	// cudaMallocManaged(), like malloc(),
	hipMallocManaged( &sum,  arraySize*sizeof(float) );				/*/*/	// returns pointers usable by both devices

	// fill first two arrays before the CUDA starts
	for (int i=0; i<arraySize; i++){
		add1[i] = 1.0;
		add2[i] = 2.0;
	}
	printf("arrays done. prepare for adding\n");

	// parallelization happens here
	addArrays<<<1,1>>>(arraySize, add1,add2,sum);						/*/*/	// <<<1,1>>> tells CPU to give task to the GPU
																		/*/*/	// the 1's will be explained later, but in
	// wait for all threads to complete on the GPU								// this case it means just one thread
	hipDeviceSynchronize();											/*/*/	// then we wait for all GPU threads to finish calculating	
	printf("adding complete.\t");												// now the CPU is back in charge

	// check for accuracy- what's the biggest mistake?
	float maxError = 0.0;
	for (int i=0; i<arraySize; i++){
		// check each array index for value and store the greater deviation from 3.0
		maxError = fmax(maxError, fabs(sum[i]-3.0));
	}
	printf("max error = %f\n",maxError);

	// free memory
	hipFree(add1);														/*/*/	// we need to use cudaFree()
	hipFree(add2);														/*/*/	// instead of delete []
	hipFree(sum);														/*/*/	// because it's shared CUDA memory

	return 0;
}
